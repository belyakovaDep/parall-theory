#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <cmath>
#include <ctime>
#include "boost/program_options.hpp"

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

namespace boo = boost::program_options;

struct CudaDeleter {
    void operator()(double* ptr) const {
        hipFree(ptr);
    }
};

struct CudaStreamDeleter {
    void operator()(hipStream_t* stream) const {
        hipStreamDestroy(*stream);
    }
};

struct CudaGraphDeleter {
    void operator()(hipGraph_t* graph) const {
        hipGraphDestroy(*graph);
    }
};

std::unique_ptr<hipStream_t, CudaStreamDeleter> createCudaStream() {
    hipStream_t* stream = new hipStream_t;
    hipError_t err = hipStreamCreate(stream);
    return std::unique_ptr<hipStream_t, CudaStreamDeleter>(stream);
}

std::unique_ptr<hipGraph_t, CudaGraphDeleter> createCudaGraph() {
    hipGraph_t* graph = new hipGraph_t;
    return std::unique_ptr<hipGraph_t, CudaGraphDeleter>(graph);
}

__global__
void calculateMatrix(double* matrixA, double* matrixB, size_t size)
{
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;

	if (i * size + j > size * size) return;
	
	if(!((j == 0 || i == 0 || j == size - 1 || i == size - 1)))
	{
		matrixB[i * size + j] = 0.25 * (matrixA[i * size + j - 1] + matrixA[(i - 1) * size + j] +
							matrixA[(i + 1) * size + j] + matrixA[i * size + j + 1]);		
	}
}

__global__
void getErrorMatrix(double* matrixA, double* matrixB, double* outputMatrix, size_t size)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > size * size) return;

	outputMatrix[idx] = std::abs(matrixB[idx] - matrixA[idx]);
}

const double corner1 = 10;
const double corner2 = 20;
const double corner3 = 30;
const double corner4 = 20;

const int error_step = 100;

int main(int argc, char** argv)
{

	boo::options_description desc{"Options"};
    desc.add_options()
      ("help,h", "help screen")
      ("accuracy", boo::value<float>()->default_value(0.000001), "the least avaliable error value")
      ("size", boo::value<int>()->default_value(10), "size of the matrix")
      ("iterations", boo::value<int>()->default_value(50), "the most avaliable number of iterations");

    boo::variables_map vm;
    boo::store(parse_command_line(argc, argv, desc), vm);
    boo::notify(vm);

    if (vm.count("help")) std::cout << desc << '\n';
    const size_t size = vm["size"].as<int>();
    const size_t maxIter = vm["iterations"].as<int>();
    const float minError = vm["accuracy"].as<float>();
    
    const size_t totalSize = size * size;
	
	std::unique_ptr<double, CudaDeleter> uMatrixA;
	double *matrixA = uMatrixA.get();

	std::unique_ptr<double, CudaDeleter> uMatrixB;
	double *matrixB = uMatrixB.get();

	hipHostMalloc(&matrixA, totalSize * sizeof(double));
	hipHostMalloc(&matrixB, totalSize * sizeof(double));
	
	std::memset(matrixA, 0, totalSize * sizeof(double));

	matrixA[0] = corner1;
	matrixA[size - 1] = corner2;
	matrixA[size * size - 1] = corner3;
	matrixA[size * (size - 1)] = corner4;

	const double step = 1.0 * (corner2 - corner1) / (size - 1);
	for (int i = 1; i < size - 1; i++)
	{
		matrixA[i] = corner1 + i * step;
		matrixA[i * size] = corner1 + i * step;
		matrixA[size - 1 + i * size] = corner2 + i * step;
		matrixA[size * (size - 1) + i] = corner4 + i * step;
	}

	std::memcpy(matrixB, matrixA, totalSize * sizeof(double));

	size_t tempStorageSize = 0;

	std::unique_ptr<double, CudaDeleter> uDeviceMatrixAPtr;
	double *deviceMatrixAPtr = uDeviceMatrixAPtr.get();

	std::unique_ptr<double, CudaDeleter> uDeviceMatrixBPtr;
	double *deviceMatrixBPtr = uDeviceMatrixBPtr.get();

	std::unique_ptr<double, CudaDeleter> uDeviceError;
	double *deviceError = uDeviceError.get();

	std::unique_ptr<double, CudaDeleter> uErrorMatrix;
	double *errorMatrix = uErrorMatrix.get();

	std::unique_ptr<double, CudaDeleter> uTempStorage;
	double *tempStorage = uTempStorage.get();

	hipError_t cudaStatus_1 = hipMalloc((void**)(&deviceMatrixAPtr), sizeof(double) * totalSize);
	hipError_t cudaStatus_2 = hipMalloc((void**)(&deviceMatrixBPtr), sizeof(double) * totalSize);
	hipMalloc((void**)&deviceError, sizeof(double));
	cudaStatus_1 = hipMalloc((void**)&errorMatrix, sizeof(double) * totalSize);

	
	if (cudaStatus_1 != 0 || cudaStatus_2 != 0)
	{
		std::cout << "Memory allocation error" << std::endl;
		return -1;
	}

	cudaStatus_1 = hipMemcpy(deviceMatrixAPtr, matrixA, sizeof(double) * totalSize, hipMemcpyHostToDevice);
	cudaStatus_2 = hipMemcpy(deviceMatrixBPtr, matrixB, sizeof(double) * totalSize, hipMemcpyHostToDevice);

	if (cudaStatus_1 != 0 || cudaStatus_2 != 0)
	{
		std::cout << "Memory transfering error" << std::endl;
		return -1;
	}

	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, totalSize);
	
	hipMalloc((void**)&tempStorage, tempStorageSize);

	int iter = 0; 
	std::unique_ptr<double, CudaDeleter> uError;
	double *error = uError.get();
	hipHostMalloc(&error, sizeof(double));
	*error = 1.0;

	auto uStream = createCudaStream();
    hipStream_t stream = *uStream.get();

	bool isGraphCreated = false;
	//hipStream_t stream;
	auto uGraph = createCudaGraph();
	auto graph = uGraph.get();
	hipGraphExec_t instance;

	size_t threads = (size < 1024) ? size : 1024;
    unsigned int blocks = size / threads;

	dim3 blockDim(32, 32);
    dim3 gridDim((size + blockDim.x - 1) /  blockDim.x, (size + blockDim.y - 1) /  blockDim.y);

    std::cout << "Start: " << std::endl;

	clock_t begin = clock();
	while(iter < maxIter && *error > minError)
	{
		if (isGraphCreated)
		{
			hipGraphLaunch(instance, stream);
			
			hipMemcpyAsync(error, deviceError, sizeof(double), hipMemcpyDeviceToHost, stream);

			hipStreamSynchronize(stream);

			iter += error_step;
		}
		else
		{
			hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
			for(size_t i = 0; i < error_step / 2; i++)
			{
				calculateMatrix<<<gridDim, blockDim, 0, stream>>>(deviceMatrixAPtr, deviceMatrixBPtr, size);
				calculateMatrix<<<gridDim, blockDim, 0, stream>>>(deviceMatrixBPtr, deviceMatrixAPtr, size);
			}

			getErrorMatrix<<<threads * blocks * blocks, threads,  0, stream>>>(deviceMatrixAPtr, deviceMatrixBPtr, errorMatrix, size);
			hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, errorMatrix, deviceError, totalSize, stream);
	
			hipStreamEndCapture(stream, graph);
			hipGraphInstantiate(&instance, *graph, NULL, NULL, 0);
			isGraphCreated = true;
  		}
	}

	clock_t end = clock();
	std::cout << "Time: " << 1.0 * (end - begin) / CLOCKS_PER_SEC << std::endl;
	std::cout << "Iter: " << iter << " Error: " << *error << std::endl;

	return 0;
}